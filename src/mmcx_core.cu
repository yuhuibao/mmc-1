#include "hip/hip_runtime.h"
/***************************************************************************/ /**
                                                                               **  \mainpage Monte Carlo eXtreme - GPU accelerated Monte Carlo Photon Migration \
                                                                               **      -- OpenCL edition
                                                                               **  \author Qianqian Fang <q.fang at neu.edu>
                                                                               **  \copyright Qianqian Fang, 2009-2018
                                                                               **
                                                                               **  \section sref Reference:
                                                                               **  \li \c (\b Yu2018) Leiming Yu, Fanny Nina-Paravecino, David Kaeli, and Qianqian Fang,
                                                                               **         "Scalable and massively parallel Monte Carlo photon transport simulations
                                                                               **         for heterogeneous computing platforms," J. Biomed. Optics, 23(1), 010504 (2018)
                                                                               **
                                                                               **  \section slicense License
                                                                               **          GPL v3, see LICENSE.txt for details
                                                                               *******************************************************************************/
#include "debug.h"
#include "mmcx_core.h"

#include "xorshift128p_rand.cu"  ///< use xorshift128+ RNG (XORSHIFT128P)

#ifdef __CUDA_ARCH__
#define __constant const
#define __private
#define __local
#define __global
#define __kernel __global__
#define get_global_id

__device__ __host__ float4 convert_float4_rte(float4 v) {
  return make_float4(roundf(v.x), roundf(v.y), roundf(v.z), roundf(v.w));
}

__device__ __host__ float3 operator*(float3 a, float3 b) {
  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
__device__ __host__ float3 operator*(float f, float3 v) {
  return make_float3(v.x * f, v.y * f, v.z * f);
}
__device__ __host__ float3 operator*(float3 v, float f) {
  return make_float3(v.x * f, v.y * f, v.z * f);
}
__device__ __host__ float4 operator*(float4 v, float f) {
  return make_float4(v.x * f, v.y * f, v.z * f, v.w * f);
}
__device__ __host__ float4 operator-(float4 v) {
  return make_float4(-v.x, -v.y, -v.z, -v.w);
}
__device__ __host__ int4 operator-(int4 v) {
  return make_int4(-v.x, -v.y, -v.z, -v.w);
}
__device__ __host__ void operator*=(float3 &b, float f) {
  b.x *= f;
  b.y *= f;
  b.z *= f;
}
__device__ __host__ float3 operator+(float3 a, float3 b) {
  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
__device__ __host__ void operator+=(float3 &b, float3 a) {
  b.x += a.x;
  b.y += a.y;
  b.z += a.z;
}
__device__ __host__ float3 operator-(float3 a, float3 b) {
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
__device__ __host__ void operator-=(float3 &b, float3 a) {
  b.x -= a.x;
  b.y -= a.y;
  b.z -= a.z;
}
__device__ float3 operator/(float3 v, float f) {
  float inv = 1.0f / f;
  return v * inv;
}
__device__ void operator/=(float3 &b, float f) {
  float inv = 1.0f / f;
  b.x *= inv;
  b.y *= inv;
  b.z *= inv;
}

__device__ float4 operator+(float4 a, float4 b) {
  return make_float4(a.x + b.x, a.y + b.y, a.z + b.w, a.w + b.w);
}
__device__ __host__ float4 isgreater(float4 a, float4 b) {
  return make_float4(a.x > b.x, a.y > b.y, a.z > b.z, a.w > b.w);
}
__device__ void operator+=(float4 &b, float4 a) {
  b.x += a.x;
  b.y += a.y;
  b.z += a.z;
  b.w += a.w;
}
__device__ float4 operator*(float4 a, float4 b) {
  return make_float4(a.x * b.x, a.y * b.y, a.z * b.w, a.w * b.w);
}
__device__ float4 operator/(float4 a, float4 b) {
  return make_float4(a.x / b.x, a.y / b.y, a.z / b.w, a.w / b.w);
}
__device__ float4 operator-(float4 a, float4 b) {
  return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w);
}
__device__ void operator-=(float4 &b, float4 a) {
  b.x -= a.x;
  b.y -= a.y;
  b.z -= a.z;
  b.w -= a.w;
}

__device__ float dot(float3 a, float3 b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float clamp(float f, float a, float b) { return max(a, min(f, b)); }

#define FL4(f) make_float4(f, f, f, f)
#define FL3(f) make_float3(f, f, f)
#define atomicadd(a, b) atomicAdd(a, b)
#define atomic_add(a, b) atomicAdd(a, b)
#define sincos(a, b) sincosf(a, b)

#else
#define FL4(f) (f)
#define FL3(f) (f)
#endif

#ifdef MCX_SAVE_DETECTORS
#pragma OPENCL EXTENSION cl_khr_global_int32_base_atomics : enable
#endif

#ifdef USE_HALF
#pragma OPENCL EXTENSION cl_khr_fp16 : enable
#define FLOAT4VEC half4
#define TOFLOAT4 convert_half4
#else
#define FLOAT4VEC float4
#define TOFLOAT4
#endif
#define CUDA_ASSERT(a)      mcx_cu_assess((a),__FILE__,__LINE__) ///< macro to report CUDA errors
#ifdef MCX_USE_NATIVE
#define MCX_MATHFUN(fun) native_##fun
#define MCX_SINCOS(theta, osin, ocos) \
  {                                   \
    (osin) = native_sin(theta);       \
    (ocos) = native_cos(theta);       \
  }
#else
#define MCX_MATHFUN(fun) fun
//#define MCX_SINCOS(theta,osin,ocos)   (ocos)=sincos((theta),&(osin))
#endif

#define R_PI 0.318309886183791f

#define ONE_PI 3.1415926535897932f  // pi
#define TWO_PI 6.28318530717959f    // 2*pi

#define C0 299792458000.f            // speed of light in mm/s
#define R_C0 3.335640951981520e-12f  // 1/C0 in s/mm

#define VERY_BIG (1.f / FLT_EPSILON)  // a big number
#define JUST_ABOVE_ONE 1.0001f        // test for boundary
#define SAME_VOXEL -9999.f            // scatter within a voxel
#define NO_LAUNCH 9999                // when fail to launch, for debug
#define MAX_PROP 2000                 /*maximum property number*/
#define ID_UNDEFINED \
  0xFFFFFFFFU /**< flag indicating the index is outside of the volume */

#define DET_MASK 0xFFFF0000
#define MED_MASK 0x0000FFFF
#define MAX_ACCUM 1000.f
#define R_MIN_MUS 1e9f
#define FIX_PHOTON 1e-3f /**< offset to the ray to avoid edge/vertex */
#define MAX_TRIAL 3 /**< number of fixes when a photon hits an edge/vertex */

#define MCX_DEBUG_MOVE 1
#define MCX_DEBUG_PROGRESS 2048

#define MMC_UNDEFINED (3.40282347e+38F)

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define F32N(a) \
  ((a)&0x80000000) /**<  Macro to test if a floating point is negative */
#define F32P(a) \
  ((a) ^ 0x80000000) /**<  Macro to test if a floating point is positive */

int ifaceorder[] = {3, 0, 2, 1};
//__constant int fc[4][3]={{0,4,2},{3,5,4},{2,5,1},{1,3,0}};
//__constant int nc[4][3]={{3,0,1},{3,1,2},{2,0,3},{1,0,2}};
int out[4][3] = {{0, 3, 1}, {3, 2, 1}, {0, 2, 3}, {0, 1, 2}};
int facemap[] = {2, 0, 1, 3};
int ifacemap[] = {1, 2, 0, 3};

#ifdef USE_ATOMIC

#ifndef __CUDA_ARCH__

// OpenCL float atomicadd hack:
// http://suhorukov.blogspot.co.uk/2011/12/opencl-11-atomic-operations-on-floating.html
// https://devtalk.nvidia.com/default/topic/458062/atomicadd-float-float-atomicmul-float-float-/

inline float atomicadd(volatile __global float *address, const float value) {
  float old = value;
  while ((old = atomic_xchg(address, atomic_xchg(address, 0.0f) + old)) != 0.0f)
    ;
  return old;
}

/*

#pragma OPENCL EXTENSION cl_khr_fp64: enable
#pragma OPENCL EXTENSION cl_khr_int64_base_atomics : enable

inline double atomicadd(__global double *val, const double delta){
  union {
  double f;
  ulong  i;
  } old, new;

  do{
     old.f = *val;
     new.f = old.f + delta;
  } while (atom_cmpxchg((volatile __global ulong *)val, old.i, new.i) != old.i);
  return old.f;
}
*/
#endif

#endif

__device__ void clearpath(float *p, int len) {
  uint i;
  for (i = 0; i < len; i++) p[i] = 0.f;
}

#ifdef MCX_SAVE_DETECTORS
__device__ uint finddetector(float3 *p0, float4 *gdetpos,
                             __constant MCXParam *gcfg) {
  uint i;
  for (i = 0; i < gcfg->detnum; i++) {
    if ((gdetpos[i].x - p0[0].x) * (gdetpos[i].x - p0[0].x) +
            (gdetpos[i].y - p0[0].y) * (gdetpos[i].y - p0[0].y) +
            (gdetpos[i].z - p0[0].z) * (gdetpos[i].z - p0[0].z) <
        gdetpos[i].w) {
      return i + 1;
    }
  }
  return 0;
}

__device__ void savedetphoton(__global float *n_det,
                              __global uint *detectedphoton, float *ppath,
                              float3 *p0, float3 *v, float4 *gdetpos,
                              int extdetid, __constant MCXParam *gcfg) {
  uint detid = (extdetid < 0) ? finddetector(p0, gdetpos, gcfg) : extdetid;
  if (detid) {
    uint baseaddr = atomic_inc(detectedphoton);
    if (baseaddr < gcfg->maxdetphoton) {
      uint i;
      baseaddr *= (gcfg->reclen + 1);
      n_det[baseaddr++] = detid;
      for (i = 0; i < (gcfg->maxmedia << 1); i++)
        n_det[baseaddr++] = ppath[i];  // save partial pathlength to the memory
      for (i = 0; i < gcfg->ismomentum * gcfg->maxmedia; i++)
        n_det[baseaddr++] =
            ppath[i + (gcfg->maxmedia
                       << 1)];  // save partial pathlength to the memory
      if (gcfg->issaveexit) {
        n_det[baseaddr++] = p0->x;
        n_det[baseaddr++] = p0->y;
        n_det[baseaddr++] = p0->z;
        n_det[baseaddr++] = v->x;
        n_det[baseaddr++] = v->y;
        n_det[baseaddr++] = v->z;
      }
      n_det[baseaddr++] =
          ppath[gcfg->reclen - 1];  // save partial pathlength to the memory
    }
  }
}
#endif

/**
 * \brief Branch-less Badouel-based SSE4 ray-tracer to advance photon by one
 * step
 *
 * this function uses Branch-less Badouel-based SSE4 ray-triangle intersection
 * tests to advance photon by one step, see Fang2012. Both Badouel and
 * Branch-less Badouel algorithms do not calculate the Barycentric coordinates
 * and can only store energy loss using 0-th order basis function. This function
 * is the fastest among the 4 ray-tracers.
 *
 * \param[in,out] r: the current ray
 * \param[in] tracer: the ray-tracer aux data structure
 * \param[in] cfg: simulation configuration structure
 * \param[out] visit: statistics counters of this thread
 */

__device__ float branchless_badouel_raytet(ray *r, __constant MCXParam *gcfg,
                                           __constant int *elem,
                                           __global float *weight, int type,
                                           __constant int *facenb,
                                           float4 *normal,
                                           __constant medium *med) {
  int faceorder[] = {1, 3, 2, 0, -1};
  float Lmin;
  float ww, totalloss = 0.f;
  int tshift, faceidx = -1, eid;
  float4 T, S;
  union {
    float f;
    uint i;
  } currweight;

  if (r->eid <= 0) return -1;

  eid = (r->eid - 1) << 2;

  r->pout.x = MMC_UNDEFINED;
  r->faceid = -1;
  r->isend = 0;

  S = FL4(r->vec.x) * normal[eid] + FL4(r->vec.y) * normal[eid + 1] +
      FL4(r->vec.z) * normal[eid + 2];
  T = normal[eid + 3] -
      (FL4(r->p0.x) * normal[eid] + FL4(r->p0.y) * normal[eid + 1] +
       FL4(r->p0.z) * normal[eid + 2]);
  T = -(isgreater(T, FL4(0.f)) * 2.f) * FL4(0.5f) * T;
  T = T / S;

  S = -convert_float4_rte(isgreater(S, FL4(0.f)) * 2.f) * FL4(0.5f);
  T = S * T + (FL4(1.f) - S) * FL4(1e10f);

  eid = r->eid - 1;

  Lmin = fmin(fmin(fmin(T.x, T.y), T.z), T.w);
  faceidx = ((Lmin == 1e10f)
                 ? 4
                 : Lmin == T.x ? 0 : (Lmin == T.y ? 1 : (Lmin == T.z ? 2 : 3)));
  r->faceid = faceorder[faceidx];

  if (r->faceid >= 0 && Lmin >= 0.f) {
    medium prop;

    prop = med[type];
    currweight.f = r->weight;

    r->Lmove = (prop.mus <= EPS) ? R_MIN_MUS : r->slen / prop.mus;
    r->isend = (Lmin > r->Lmove);
    r->Lmove = ((r->isend) ? r->Lmove : Lmin);
    r->pout = r->p0 + FL3(Lmin) * r->vec;

    if ((int)((r->photontimer + r->Lmove * (prop.n * R_C0) - gcfg->tstart) *
              gcfg->Rtstep) > gcfg->maxgate - 1) { /*exit time window*/
      r->faceid = -2;
      r->pout.x = MMC_UNDEFINED;
      r->Lmove = (gcfg->tend - r->photontimer) / (prop.n * R_C0) - 1e-4f;
    }
    totalloss = MCX_MATHFUN(exp)(-prop.mua * r->Lmove);
    r->weight *= totalloss;

    totalloss = 1.f - totalloss; /*remaining fraction*/
    r->slen -= r->Lmove * prop.mus;
    ww = currweight.f - r->weight;
    r->photontimer += r->Lmove * (prop.n * R_C0);
    /*
                if(gcfg->outputtype==otWL || gcfg->outputtype==otWP)
                        tshift=MIN(
       ((int)(replaytime[r->photonid]*gcfg->Rtstep)), gcfg->maxgate-1
       )*gcfg->framelen; else
    */
    tshift = MIN(((int)((r->photontimer - gcfg->tstart) * gcfg->Rtstep)),
                 gcfg->maxgate - 1) *
             gcfg->framelen;
    {
#ifndef MCX_SKIP_VOLUME
      if (prop.mua > 0.f) {
        if (gcfg->outputtype != otEnergy && gcfg->outputtype != otWP)
          ww /= prop.mua;
      }
#ifndef USE_DMMC
      uint newidx = eid + tshift;
      r->oldidx = (r->oldidx == ID_UNDEFINED) ? newidx : r->oldidx;
      if (newidx != r->oldidx) {
#ifndef DO_NOT_SAVE
#ifdef USE_ATOMIC
        atomicadd(weight + r->oldidx, r->oldweight);
#else
        weight[r->oldidx] += r->oldweight;
#endif
#endif
        r->oldidx = newidx;
        r->oldweight = ww;
      } else {
        r->oldweight += ww;
      }
#else
      eid = (int)(r->Lmove * gcfg->dstep) + 1;  // number of segments
      eid = (eid << 1);
      S.w = r->Lmove / eid;                               // segment length
      T.w = MCX_MATHFUN(exp)(-prop.mua * S.w);            // segment loss
      T.xyz = r->vec * FL3(S.w);                          // delta vector
      S.xyz = (r->p0 - gcfg->nmin) + (T.xyz * FL3(0.5f)); /*starting point*/
      totalloss =
          (totalloss == 0.f)
              ? 0.f
              : (1.f - T.w) / totalloss;  // fraction of total loss per segment
      S.w = ww;                           // S.w is now the current weight
      for (faceidx = 0; faceidx < eid; faceidx++) {
        int3 idx = convert_int3_rtn(S.xyz * FL3(gcfg->dstep));
        idx = idx & (idx >= (int3)(0));
        uint newidx =
            (idx.z * gcfg->crop0.y + idx.y * gcfg->crop0.x + idx.x) + tshift;
        r->oldidx = (r->oldidx == ID_UNDEFINED) ? newidx : r->oldidx;
        if (newidx != r->oldidx) {
#ifndef DO_NOT_SAVE
#ifdef USE_ATOMIC
          atomicadd(weight + r->oldidx, r->oldweight);
#else
          weight[r->oldidx] += r->oldweight;
#endif
#endif
          r->oldidx = newidx;
          r->oldweight = S.w * totalloss;
        } else
          r->oldweight += S.w * totalloss;
        S.w *= T.w;
        S.xyz += T.xyz;
      }
#endif
#endif
    }
    r->p0 = r->p0 + FL3(r->Lmove) * r->vec;
  }
  return ((r->faceid == -2) ? 0.f : r->slen);
}

/**
 * @brief Calculate the reflection/transmission of a ray at an interface
 *
 * This function handles the reflection and transmission events at an interface
 * where the refractive indices mismatch.
 *
 * \param[in,out] cfg: simulation configuration structure
 * \param[in] c0: the current direction vector of the ray
 * \param[in] tracer: the ray-tracer aux data structure
 * \param[in] oldeid: the index of the element the photon moves away from
 * \param[in] eid: the index of the element the photon about to move into
 * \param[in] faceid: index of the face through which the photon
 * reflects/transmits \param[in,out] ran: the random number generator states
 */

#ifdef MCX_DO_REFLECTION

__device__ float reflectray(__constant MCXParam *gcfg, float3 *c0, int *oldeid,
                            int *eid, int faceid, __private GRandType *ran,
                            __constant int *type, float4 *normal,
                            __constant medium *med) {
  /*to handle refractive index mismatch*/
  float3 pnorm = {0.f, 0.f, 0.f};
  float Icos, Re, Im, Rtotal, tmp0, tmp1, tmp2, n1, n2;
  int offs = (*oldeid - 1) << 2;

  faceid = ifaceorder[faceid];
  /*calculate the normal direction of the intersecting triangle*/
  pnorm.x = ((__constant float *)&(normal[offs]))[faceid];
  pnorm.y = ((__constant float *)&(normal[offs]))[faceid + 4];
  pnorm.z = ((__constant float *)&(normal[offs]))[faceid + 8];

  /*pn pointing outward*/

  /*compute the cos of the incidence angle*/
  Icos = fabs(dot(*c0, pnorm));

  n1 = ((*oldeid != *eid) ? med[type[*oldeid - 1]].n : gcfg->nout);
  n2 = ((*eid > 0) ? med[type[*eid - 1]].n : gcfg->nout);

  tmp0 = n1 * n1;
  tmp1 = n2 * n2;
  tmp2 = 1.f -
         tmp0 / tmp1 * (1.f - Icos * Icos); /*1-[n1/n2*sin(si)]^2 = cos(ti)^2*/

  if (tmp2 > 0.f) {                        /*if no total internal reflection*/
    Re = tmp0 * Icos * Icos + tmp1 * tmp2; /*transmission angle*/
    tmp2 = MCX_MATHFUN(sqrt)(tmp2);        /*to save one sqrt*/
    Im = 2.f * n1 * n2 * Icos * tmp2;
    Rtotal = (Re - Im) / (Re + Im); /*Rp*/
    Re = tmp1 * Icos * Icos + tmp0 * tmp2 * tmp2;
    Rtotal = (Rtotal + (Re - Im) / (Re + Im)) * 0.5f; /*(Rp+Rs)/2*/
    if (*oldeid == *eid) return Rtotal;     /*initial specular reflection*/
    if (rand_next_reflect(ran) <= Rtotal) { /*do reflection*/
      *c0 += (FL3(-2.f * Icos)) * pnorm;
      // if(gcfg->debuglevel&dlReflect) MMC_FPRINTF(("R %f %f %f %d %d
      // %f\n",c0->x,c0->y,c0->z,*eid,*oldeid,Rtotal));
      *eid = *oldeid; /*stay with the current element*/
    } else if (gcfg->isspecular == 2 && *eid == 0) {
      // if do transmission, but next neighbor is 0, terminate
    } else { /*do transmission*/
      *c0 += (FL3(-Icos)) * pnorm;
      *c0 = (FL3(tmp2)) * pnorm + FL3(n1 / n2) * (*c0);
      // if(gcfg->debuglevel&dlReflect) MMC_FPRINTF(("Z %f %f %f %d %d
      // %f\n",c0->x,c0->y,c0->z,*eid,*oldeid,1.f-Rtotal));
    }
  } else { /*total internal reflection*/
    *c0 += (FL3(-2.f * Icos)) * pnorm;
    *eid = *oldeid;
    // if(gcfg->debuglevel&dlReflect) MMC_FPRINTF(("V %f %f %f %d %d
    // %f\n",c0->x,c0->y,c0->z,*eid,*oldeid,1.f));
  }
  tmp0 = MCX_MATHFUN(rsqrt)(dot(*c0, *c0));
  (*c0) *= FL3(tmp0);
  return 1.f;
}

#endif

/**
 * @brief Performing one scattering event of the photon
 *
 * This function updates the direction of the photon by performing a scattering
 * calculation
 *
 * @param[in] g: anisotropy g
 * @param[out] dir: current ray direction vector
 * @param[out] ran: random number generator states
 * @param[out] cfg: the simulation configuration
 * @param[out] pmom: buffer to store momentum transfer data if needed
 */

__device__ float mc_next_scatter(float g, float3 *dir, __private GRandType *ran,
                                 __constant MCXParam *gcfg, float *pmom) {
  float nextslen;
  float sphi, cphi, tmp0, theta, stheta, ctheta, tmp1;
  float3 p;

  // random scattering length (normalized)
  nextslen = rand_next_scatlen(ran);

  tmp0 = TWO_PI * rand_next_aangle(ran);  // next arimuth angle
  sincosf(tmp0, &sphi, &cphi);

  if (g > EPS) {  // if g is too small, the distribution of theta is bad
    tmp0 = (1.f - g * g) / (1.f - g + 2.f * g * rand_next_zangle(ran));
    tmp0 *= tmp0;
    tmp0 = (1.f + g * g - tmp0) / (2.f * g);
    tmp0 = clamp(tmp0, -1.f, 1.f);

    theta = acos(tmp0);
    stheta = MCX_MATHFUN(sqrt)(1.f - tmp0 * tmp0);
    // stheta=MCX_MATHFUN(sin)(theta);
    ctheta = tmp0;
  } else {
    theta = acos(2.f * rand_next_zangle(ran) - 1.f);
    sincosf(theta, &stheta, &ctheta);
  }

  if (dir->z > -1.f + EPS && dir->z < 1.f - EPS) {
    tmp0 = 1.f - dir->z * dir->z;  // reuse tmp to minimize registers
    tmp1 = MCX_MATHFUN(rsqrt)(tmp0);
    tmp1 = stheta * tmp1;

    p.x = tmp1 * (dir->x * dir->z * cphi - dir->y * sphi) + dir->x * ctheta;
    p.y = tmp1 * (dir->y * dir->z * cphi + dir->x * sphi) + dir->y * ctheta;
    p.z = -tmp1 * tmp0 * cphi + dir->z * ctheta;
  } else {
    p.x = stheta * cphi;
    p.y = stheta * sphi;
    p.z = (dir->z > 0.f) ? ctheta : -ctheta;
  }
  if (gcfg->ismomentum) pmom[0] += (1.f - ctheta);

  dir->x = p.x;
  dir->y = p.y;
  dir->z = p.z;
  return nextslen;
}

/**
 * \brief Function to deal with ray-edge/ray-vertex intersections
 *
 * when a photon is crossing a vertex or edge, (slightly) pull the
 * photon toward the center of the element and try again
 *
 * \param[in,out] p: current photon position
 * \param[in] nodes: pointer to the 4 nodes of the tet
 * \param[in] ee: indices of the 4 nodes ee=elem[eid]
 */

__device__ void fixphoton(float3 *p, __global float3 *nodes,
                          __constant int *ee) {
  float3 c0 = {0.f, 0.f, 0.f};
  int i;
  /*calculate element centroid*/
  for (i = 0; i < 4; i++) c0 += nodes[ee[i] - 1];
  *p += (c0 * FL3(0.25f) - *p) * (FL3(FIX_PHOTON));
}

/**
 * @brief Launch a new photon
 *
 * This function launch a new photon using one of the dozen supported source
 * forms.
 *
 * \param[in,out] cfg: simulation configuration structure
 * \param[in,out] r: the current ray
 * \param[in] mesh: the mesh data structure
 * \param[in,out] ran: the random number generator states
 */

__device__ void launchphoton(__constant MCXParam *gcfg, ray *r,
                             __global float3 *node, __constant int *elem,
                             __constant int *srcelem,
                             __private GRandType *ran) {
  int canfocus = 1;
  float3 origin = r->p0;

  r->slen = rand_next_scatlen(ran);
#if defined(MCX_SRC_PENCIL)
  if (r->eid > 0) return;
#elif defined(MCX_SRC_PLANAR) || defined(MCX_SRC_PATTERN) || \
    defined(MCX_SRC_PATTERN3D) ||                            \
    defined(MCX_SRC_FOURIER) /*a rectangular grid over a plane*/
  float rx = rand_uniform01(ran);
  float ry = rand_uniform01(ran);
  r->p0.x = gcfg->srcpos.x + rx * gcfg->srcparam1.x + ry * gcfg->srcparam2.x;
  r->p0.y = gcfg->srcpos.y + rx * gcfg->srcparam1.y + ry * gcfg->srcparam2.y;
  r->p0.z = gcfg->srcpos.z + rx * gcfg->srcparam1.z + ry * gcfg->srcparam2.z;
  r->weight = 1.f;
#if defined(MCX_SRC_PATTERN)
  if (gcfg->srctype == stPattern) {
    int xsize = (int)gcfg->srcparam1.w;
    int ysize = (int)gcfg->srcparam2.w;
    // r->posidx=MIN((int)(ry*ysize),ysize-1)*xsize+MIN((int)(rx*xsize),xsize-1);
#elif defined(MCX_SRC_FOURIER)  // need to prevent rx/ry=1 here
  r->weight = (MCX_MATHFUN(cos)((floor(gcfg->srcparam1.w) * rx +
                                 floor(gcfg->srcparam2.w) * ry +
                                 gcfg->srcparam1.w - floor(gcfg->srcparam1.w)) *
                                TWO_PI) *
                   (1.f - gcfg->srcparam2.w + floor(gcfg->srcparam2.w)) +
               1.f) *
              0.5f;
#endif
    origin.x += (gcfg->srcparam1.x + gcfg->srcparam2.x) * 0.5f;
    origin.y += (gcfg->srcparam1.y + gcfg->srcparam2.y) * 0.5f;
    origin.z += (gcfg->srcparam1.z + gcfg->srcparam2.z) * 0.5f;
#elif defined(MCX_SRC_FOURIERX) || \
    defined(MCX_SRC_FOURIERX2D)  // [v1x][v1y][v1z][|v2|]; [kx][ky][phi0][M],
                                 // unit(v0) x unit(v1)=unit(v2)
  float rx = rand_uniform01(ran);
  float ry = rand_uniform01(ran);
  float4 v2 = gcfg->srcparam1;
  v2.w *= MCX_MATHFUN(rsqrt)(gcfg->srcparam1.x * gcfg->srcparam1.x +
                             gcfg->srcparam1.y * gcfg->srcparam1.y +
                             gcfg->srcparam1.z * gcfg->srcparam1.z);
  v2.x = v2.w * (gcfg->srcdir.y * gcfg->srcparam1.z -
                 gcfg->srcdir.z * gcfg->srcparam1.y);
  v2.y = v2.w * (gcfg->srcdir.z * gcfg->srcparam1.x -
                 gcfg->srcdir.x * gcfg->srcparam1.z);
  v2.z = v2.w * (gcfg->srcdir.x * gcfg->srcparam1.y -
                 gcfg->srcdir.y * gcfg->srcparam1.x);
  r->p0.x = gcfg->srcpos.x + rx * gcfg->srcparam1.x + ry * v2.x;
  r->p0.y = gcfg->srcpos.y + rx * gcfg->srcparam1.y + ry * v2.y;
  r->p0.z = gcfg->srcpos.z + rx * gcfg->srcparam1.z + ry * v2.z;
#if defined(MCX_SRC_FOURIERX2D)
  r->weight =
      (MCX_MATHFUN(sin)((gcfg->srcparam2.x * rx + gcfg->srcparam2.z) * TWO_PI) *
           MCX_MATHFUN(sin)((gcfg->srcparam2.y * ry + gcfg->srcparam2.w) *
                            TWO_PI) +
       1.f) *
      0.5f;  // between 0 and 1
#else
  r->weight = (MCX_MATHFUN(cos)((gcfg->srcparam2.x * rx +
                                 gcfg->srcparam2.y * ry + gcfg->srcparam2.z) *
                                TWO_PI) *
                   (1.f - gcfg->srcparam2.w) +
               1.f) *
              0.5f;  // between 0 and 1
#endif
  origin.x += (gcfg->srcparam1.x + v2.x) * 0.5f;
  origin.y += (gcfg->srcparam1.y + v2.y) * 0.5f;
  origin.z += (gcfg->srcparam1.z + v2.z) * 0.5f;
#elif defined(MCX_SRC_DISK) || \
    defined(MCX_SRC_GAUSSIAN)  // uniform disk distribution or Gaussian-beam
  float sphi, cphi;
  float phi = TWO_PI * rand_uniform01(ran);
  sphi = MCX_MATHFUN(sin)(phi);
  cphi = MCX_MATHFUN(cos)(phi);
  float r0;
#if defined(MCX_SRC_DISK)
  r0 = MCX_MATHFUN(sqrt)(rand_uniform01(ran)) * gcfg->srcparam1.x;
#else
  if (fabs(gcfg->focus) < 1e-5f || fabs(gcfg->srcparam1.y) < 1e-5f)
                    r0=MCX_MATHFUN(sqrt)(-MCX_MATHFUN(log)((rand_uniform01(ran)))*gcfg->srcparam1.x;
		else{
      float z0 = gcfg->srcparam1.x * gcfg->srcparam1.x * M_PI /
                 gcfg->srcparam1.y;  // Rayleigh range
                    r0=MCX_MATHFUN(sqrt)(-MCX_MATHFUN(log)((rand_uniform01(ran))*(1.f+(gcfg->focus*gcfg->focus/(z0*z0))))*gcfg->srcparam1.x;
		}
#endif

  if (gcfg->srcdir.z > -1.f + EPS && gcfg->srcdir.z < 1.f - EPS) {
    float tmp0 = 1.f - gcfg->srcdir.z * gcfg->srcdir.z;
    float tmp1 = r0 * MCX_MATHFUN(rsqrt)(tmp0);
    r->p0.x = gcfg->srcpos.x + tmp1 * (gcfg->srcdir.x * gcfg->srcdir.z * cphi -
                                       gcfg->srcdir.y * sphi);
    r->p0.y = gcfg->srcpos.y + tmp1 * (gcfg->srcdir.y * gcfg->srcdir.z * cphi +
                                       gcfg->srcdir.x * sphi);
    r->p0.z = gcfg->srcpos.z - tmp1 * tmp0 * cphi;
  } else {
    r->p0.x += r0 * cphi;
    r->p0.y += r0 * sphi;
  }
#elif defined(MCX_SRC_CONE) || defined(MCX_SRC_ISOTROPIC) || \
    defined(MCX_SRC_ARCSINE)
  float ang, stheta, ctheta, sphi, cphi;
  ang = TWO_PI * rand_uniform01(ran);  // next arimuth angle
  sphi = MCX_MATHFUN(sin)(ang);
  cphi = MCX_MATHFUN(cos)(ang);
#if defined(MCX_SRC_CONE)  // a solid-angle section of a uniform sphere
  do {
    ang = (gcfg->srcparam1.y > 0)
              ? TWO_PI * rand_uniform01(ran)
              : acos(2.f * rand_uniform01(ran) - 1.f);  // sine distribution
  } while (ang > gcfg->srcparam1.x);
#else
  if (gcfg->srctype == stIsotropic)               // uniform sphere
    ang = acos(2.f * rand_uniform01(ran) - 1.f);  // sine distribution
  else
    ang = M_PI *
          rand_uniform01(ran);  // uniform distribution in zenith angle, arcsine
#endif
  stheta = MCX_MATHFUN(sin)(ang);
  ctheta = MCX_MATHFUN(cos)(ang);
  r->vec.x = stheta * cphi;
  r->vec.y = stheta * sphi;
  r->vec.z = ctheta;
  canfocus = 0;
  if (gcfg->srctype == stIsotropic)
    if (r->eid > 0) return;
#elif defined(MCX_SRC_ZGAUSSIAN)
  float ang, stheta, ctheta, sphi, cphi;
  ang = TWO_PI * rand_uniform01(ran);  // next arimuth angle
  sphi = MCX_MATHFUN(sin)(ang);
  cphi = MCX_MATHFUN(cos)(ang);
                ang=MCX_MATHFUN(sqrt)(-2.f*MCX_MATHFUN(log)((rand_uniform01(ran)))*(1.f-2.f*rand_uniform01(ran))*gcfg->srcparam1.x;
		stheta=MCX_MATHFUN(sin)(ang);
		ctheta=MCX_MATHFUN(cos)(ang);
		r->vec.x=stheta*cphi;
		r->vec.y=stheta*sphi;
		r->vec.z=ctheta;
		canfocus=0;
#elif defined(MCX_SRC_LINE) || defined(MCX_SRC_SLIT)
  float t = rand_uniform01(ran);
  r->p0.x += t * gcfg->srcparam1.x;
  r->p0.y += t * gcfg->srcparam1.y;
  r->p0.z += t * gcfg->srcparam1.z;

#if defined(MCX_SRC_LINE)
  float s, p;
  t = 1.f - 2.f * rand_uniform01(ran);
  s = 1.f - 2.f * rand_uniform01(ran);
  p = MCX_MATHFUN(sqrt)(1.f - r->vec.x * r->vec.x - r->vec.y * r->vec.y) *
      (rand_uniform01(ran) > 0.5f ? 1.f : -1.f);
  float3 vv;
  vv.x = r->vec.y * p - r->vec.z * s;
  vv.y = r->vec.z * t - r->vec.x * p;
  vv.z = r->vec.x * s - r->vec.y * t;
  r->vec = vv;
  //*((float3*)&(r->vec))=(float3)(r->vec.y*p-r->vec.z*s,r->vec.z*t-r->vec.x*p,r->vec.x*s-r->vec.y*t);
#endif
  origin.x += (gcfg->srcparam1.x) * 0.5f;
  origin.y += (gcfg->srcparam1.y) * 0.5f;
  origin.z += (gcfg->srcparam1.z) * 0.5f;
  canfocus = (gcfg->srctype == stSlit);
#endif

  if (canfocus &&
      gcfg->focus !=
          0.f) {  // if beam focus is set, determine the incident angle
    float Rn2;
    origin.x += gcfg->focus * r->vec.x;
    origin.y += gcfg->focus * r->vec.y;
    origin.z += gcfg->focus * r->vec.z;
    if (gcfg->focus < 0.f) {  // diverging beam
      r->vec.x = r->p0.x - origin.x;
      r->vec.y = r->p0.y - origin.y;
      r->vec.z = r->p0.z - origin.z;
    } else {  // converging beam
      r->vec.x = origin.x - r->p0.x;
      r->vec.y = origin.y - r->p0.y;
      r->vec.z = origin.z - r->p0.z;
    }
    Rn2 = MCX_MATHFUN(rsqrt)(dot(r->vec, r->vec));  // normalize
    r->vec = r->vec * Rn2;
  }

  r->p0 += r->vec * EPS;

#if defined(MCX_SRC_PLANAR) || defined(MCX_SRC_PATTERN) ||    \
    defined(MCX_SRC_PATTERN3D) || defined(MCX_SRC_FOURIER) || \
    defined(MCX_SRC_FOURIERX) || defined(MCX_SRC_FOURIERX2D)
  /*Caluclate intial element id and bary-centric coordinates for area sources -
   * position changes everytime*/
  float3 vecS = FL3(0.f), vecAB, vecAC, vecN;
  int is, i, ea, eb, ec;
  float bary[4] = {0.f};
  for (is = 0; is < gcfg->srcelemlen; is++) {
    int include = 1;
    constant int *elems = elem + (srcelem[is] - 1) * gcfg->elemlen;
    for (i = 0; i < 4; i++) {
      ea = elems[out[i][0]] - 1;
      eb = elems[out[i][1]] - 1;
      ec = elems[out[i][2]] - 1;
      vecAB = node[eb] - node[ea];
      vecAC = node[ec] - node[ea];
      vecS = r->p0 - node[ea];
      vecN = cross(vecAB, vecAC);
      bary[facemap[i]] = -dot(vecS, vecN);
    }
    for (i = 0; i < 4; i++) {
      if (bary[i] < -1e-4f) {
        include = 0;
      }
    }
    if (include) {
      r->eid = srcelem[is];
      float s = 0.f;
      for (i = 0; i < 4; i++) {
        s += bary[i];
      }
      for (i = 0; i < 4; i++) {
        if ((bary[i] / s) < 1e-4f) r->faceid = ifacemap[i] + 1;
      }
      break;
    }
  }
#endif
}

/**
 * @brief The core Monte Carlo function simulating a single photon
 * (!!!Important!!!)
 *
 * This is the core Monte Carlo simulation function. It simulates the life-time
 * of a single photon packet, from launching to termination.
 *
 * \param[in] id: the linear index of the current photon, starting from 0.
 * \param[in] tracer: the ray-tracer aux data structure
 * \param[in] mesh: the mesh data structure
 * \param[in,out] ran: the random number generator states
 * \param[in,out] cfg: simulation configuration structure
 * \param[out] visit: statistics counters of this thread
 */

__device__ void onephoton(unsigned int id, float *ppath,
                          __constant MCXParam *gcfg, __global float3 *node,
                          __constant int *elem, __global float *weight,
                          __global float *dref, __constant int *type,
                          __constant int *facenb, __constant int *srcelem,
                          float4 *normal, __constant medium *med,
                          __global float *n_det, __global uint *detectedphoton,
                          float *energytot, float *energyesc, float4 *gdetpos,
                          __private GRandType *ran, int *raytet) {
  int oldeid, fixcount = 0;
  ray r = {gcfg->srcpos,
           gcfg->srcdir,
           {MMC_UNDEFINED, 0.f, 0.f},
           gcfg->e0,
           0,
           0,
           1.f,
           0.f,
           0.f,
           0.f,
           ID_UNDEFINED,
           0.f};

  // r.photonid=id;

  /*initialize the photon parameters*/
  launchphoton(gcfg, &r, node, elem, srcelem, ran);
  *energytot += r.weight;
#ifdef MCX_SAVE_DETECTORS
  if (gcfg->issavedet)
    ppath[gcfg->reclen - 1] =
        r.weight; /*last record in partialpath is the initial photon weight*/
#endif
  /*use Kahan summation to accumulate weight, otherwise, counter stops at
   * 16777216*/
  /*http://stackoverflow.com/questions/2148149/how-to-sum-a-large-number-of-float-number*/

  while (1) { /*propagate a photon until exit*/
    r.slen = branchless_badouel_raytet(&r, gcfg, elem, weight, type[r.eid - 1],
                                       facenb, normal, med);
    (*raytet)++;
    if (r.pout.x == MMC_UNDEFINED) {
      if (r.faceid == -2) break; /*reaches the time limit*/
      if (fixcount++ < MAX_TRIAL) {
        fixphoton(&r.p0, node,
                  (__constant int *)(elem + (r.eid - 1) * gcfg->elemlen));
        continue;
      }
      r.eid = ID_UNDEFINED;
      r.faceid = -1;
    }
#ifdef MCX_SAVE_DETECTORS
    if (gcfg->issavedet && r.Lmove > 0.f && type[r.eid - 1] > 0)
      ppath[gcfg->maxmedia + type[r.eid - 1] - 1] +=
          r.Lmove; /*second medianum block is the partial path*/
#endif
    /*move a photon until the end of the current scattering path*/
    while (r.faceid >= 0 && !r.isend) {
      r.p0 = r.pout;

      oldeid = r.eid;
      r.eid =
          ((__constant int *)(facenb + (r.eid - 1) * gcfg->elemlen))[r.faceid];
#ifdef MCX_DO_REFLECTION
      if (gcfg->isreflect &&
          (r.eid <= 0 ||
           (r.eid > 0 && med[type[r.eid - 1]].n != med[type[oldeid - 1]].n))) {
        if (!(r.eid <= 0 && med[type[oldeid - 1]].n == gcfg->nout))
          reflectray(gcfg, &r.vec, &oldeid, &r.eid, r.faceid, ran, type, normal,
                     med);
      }
#endif
      if (r.eid <= 0) break;
      /*when a photon enters the domain from the background*/
      if (type[oldeid - 1] == 0 && type[r.eid - 1]) {
        // if(gcfg->debuglevel&dlExit)
        MMC_FPRINTF(("e %f %f %f %f %f %f %f %d\n", r.p0.x, r.p0.y, r.p0.z,
                     r.vec.x, r.vec.y, r.vec.z, r.weight, r.eid));
        if (!gcfg->voidtime) r.photontimer = 0.f;
      }
      /*when a photon exits the domain into the background*/
      if (type[oldeid - 1] && type[r.eid - 1] == 0) {
        // if(gcfg->debuglevel&dlExit)
        MMC_FPRINTF(("x %f %f %f %f %f %f %f %d\n", r.p0.x, r.p0.y, r.p0.z,
                     r.vec.x, r.vec.y, r.vec.z, r.weight, r.eid));
        if (!gcfg->isextdet) {
          r.eid = 0;
          break;
        }
      }
      //		    if(r.eid==0 && med[type[oldeid-1]].n == gcfg->nout )
      // break;
      if (r.pout.x != MMC_UNDEFINED)  // && (gcfg->debuglevel&dlMove))
        MMC_FPRINTF(("P %f %f %f %d %u %e\n", r.pout.x, r.pout.y, r.pout.z,
                     r.eid, id, r.slen));

      r.slen = branchless_badouel_raytet(&r, gcfg, elem, weight,
                                         type[r.eid - 1], facenb, normal, med);
      (*raytet)++;
#ifdef MCX_SAVE_DETECTORS
      if (gcfg->issavedet && r.Lmove > 0.f && type[r.eid - 1] > 0)
        ppath[gcfg->maxmedia + type[r.eid - 1] - 1] += r.Lmove;
#endif
      if (r.faceid == -2) break;
      fixcount = 0;
      while (r.pout.x == MMC_UNDEFINED && fixcount++ < MAX_TRIAL) {
        fixphoton(&r.p0, node,
                  (__constant int *)(elem + (r.eid - 1) * gcfg->elemlen));
        r.slen = branchless_badouel_raytet(
            &r, gcfg, elem, weight, type[r.eid - 1], facenb, normal, med);
        (*raytet)++;
#ifdef MCX_SAVE_DETECTORS
        if (gcfg->issavedet && r.Lmove > 0.f && type[r.eid - 1] > 0)
          ppath[gcfg->maxmedia + type[r.eid - 1] - 1] += r.Lmove;
#endif
      }
      if (r.pout.x == MMC_UNDEFINED) {
        /*possibily hit an edge or miss*/
        r.eid = ID_UNDEFINED;
        break;
      }
    }
    if (r.eid <= 0 || r.pout.x == MMC_UNDEFINED) {
      // if(r.eid==0 && (gcfg->debuglevel&dlMove))
      MMC_FPRINTF(
          ("B %f %f %f %d %u %e\n", r.p0.x, r.p0.y, r.p0.z, r.eid, id, r.slen));
      if (r.eid != ID_UNDEFINED) {
        // if(gcfg->debuglevel&dlExit)
        MMC_FPRINTF(("E %f %f %f %f %f %f %f %d\n", r.p0.x, r.p0.y, r.p0.z,
                     r.vec.x, r.vec.y, r.vec.z, r.weight, r.eid));
#ifdef MCX_SAVE_DETECTORS
        if (gcfg->issavedet &&
            gcfg->issaveexit) { /*when issaveexit is set to 1*/
          copystate(ppath + (gcfg->reclen - 7), (float *)&(r.p0),
                    3); /*columns 7-5 from the right store the exit positions*/
          copystate(ppath + (gcfg->reclen - 4), (float *)&(r.vec),
                    3); /*columns 4-2 from the right store the exit dirs*/
        }
#endif
#ifdef MCX_SAVE_DREF
        if (gcfg->issaveref && r.eid < 0 && dref) {
          int tshift =
              MIN(((int)((r.photontimer - gcfg->tstart) * gcfg->Rtstep)),
                  gcfg->maxgate - 1) *
              gcfg->nf;
          dref[((-r.eid) - 1) + tshift] += r.weight;
        }
#endif
      } else if (r.faceid == -2 && (gcfg->debuglevel & dlMove)) {
        MMC_FPRINTF(("T %f %f %f %d %u %e\n", r.p0.x, r.p0.y, r.p0.z, r.eid, id,
                     r.slen));
      } else if (r.eid && r.faceid != -2 && gcfg->debuglevel & dlEdge) {
        MMC_FPRINTF(("X %f %f %f %d %u %e\n", r.p0.x, r.p0.y, r.p0.z, r.eid, id,
                     r.slen));
      }
#ifdef MCX_SAVE_DETECTORS
      if (r.eid < 0) {
        if (gcfg->isextdet && type[oldeid - 1] == gcfg->maxmedia + 1) {
          savedetphoton(n_det, detectedphoton, ppath, &(r.p0), &(r.vec),
                        gdetpos, oldeid, gcfg);
        } else {
          savedetphoton(n_det, detectedphoton, ppath, &(r.p0), &(r.vec),
                        gdetpos, -1, gcfg);
        }
        clearpath(ppath, gcfg->reclen);
      }
#endif
      break; /*photon exits boundary*/
    }
    // if(gcfg->debuglevel&dlMove)
    MMC_FPRINTF(
        ("M %f %f %f %d %u %e\n", r.p0.x, r.p0.y, r.p0.z, r.eid, id, r.slen));
    if (gcfg->minenergy > 0.f && r.weight < gcfg->minenergy &&
        (gcfg->tend - gcfg->tstart) * gcfg->Rtstep <=
            1.f) { /*Russian Roulette*/
      if (rand_do_roulette(ran) * gcfg->roulettesize <= 1.f) {
        r.weight *= gcfg->roulettesize;
        // if(gcfg->debuglevel&dlWeight)
        MMC_FPRINTF(("Russian Roulette bumps r.weight to %f\n", r.weight));
      } else
        break;
    }
    float mom = 0.f;
    r.slen = mc_next_scatter(med[type[r.eid - 1]].g, &r.vec, ran, gcfg, &mom);
#ifdef MCX_SAVE_DETECTORS
    if (gcfg->ismomentum && type[r.eid - 1] > 0) /*when ismomentum is set to 1*/
      ppath[(gcfg->maxmedia << 1) + type[r.eid - 1] - 1] +=
          mom; /*the third medianum block stores the momentum transfer*/
    if (gcfg->issavedet)
      ppath[type[r.eid - 1] - 1] +=
          1.f; /*the first medianum block stores the scattering event counts*/
#endif
  }
  *energyesc += r.weight;
}

__global__ void mmc_main_loop(const int nphoton, const int ophoton,
                              MCXParam *gcfg, float3 *node, int *elem,
                              float *weight, float *dref, int *type,
                              int *facenb, int *srcelem, float4 *normal,
                              medium *med, float4 *gdetpos, float *n_det,
                              uint *detectedphoton, uint *n_seed, int *progress,
                              float *energy, MCXReporter *reporter) {
  extern __shared__ float sharedmem[];
  GRandType t[RAND_BUF_LEN];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_rng_init(t, n_seed, idx);
  float energyesc = 0.f, energytot = 0.f;
  int raytet = 0;

  /*launch photons*/
  for (int i = 0; i < nphoton + (idx < ophoton); i++) {
    onephoton(idx * nphoton + MIN(idx, ophoton) + i,
              sharedmem + threadIdx.x * gcfg->reclen, gcfg, node, elem, weight,
              dref, type, facenb, srcelem, normal, med, n_det, detectedphoton,
              &energytot, &energyesc, gdetpos, t, &raytet);
  }
  energy[idx << 1] = energyesc;
  energy[1 + (idx << 1)] = energytot;

  //        if(gcfg->debuglevel & MCX_DEBUG_PROGRESS)
  //	    CUDA_ASSERT(hipEventCreate(&updateprogress));
  atomicadd(&(reporter->raytet), raytet);
}
