#include "hip/hip_runtime.h"
/**
 **  \mainpage Mesh-based Monte Carlo (MMC) - a 3D photon simulator
 **
 **  \author Qianqian Fang <q.fang at neu.edu>
 **
 **  \section sref Reference:
 **  \li \c (\b Fang2010) Qianqian Fang, <a
 *href="http://www.opticsinfobase.org/abstract.cfm?uri=boe-1-1-165">
 **          "Mesh-based Monte Carlo Method Using Fast Ray-Tracing
 **          in Pluker Coordinates,"</a> Biomed. Opt. Express, 1(1) 165-175
 *(2010).
 **  \li \c (\b Fang2009) Qianqian Fang and David A. Boas,
 **          <a
 *href="http://www.opticsinfobase.org/abstract.cfm?uri=oe-17-22-20178">
 **          "Monte Carlo Simulation of Photon Migration in 3D Turbid Media
 *Accelerated
 **          by Graphics Processing Units,"</a> Optics Express, 17(22)
 *20178-20190 (2009).
 **
 **  \section slicense License
 **          GPL v3, see LICENSE.txt for details
 *******************************************************************************/

/**
  \file    mmc_host.c
  \brief   << Driver program of MMC >>
*/

#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdint.h>
#include "mcx_const.h"
#include "mcx_utils.h"
#include "mmcx_core.h"
#include "mmcx_host.h"
#include "tictoc.h"
#include "debug.h"
#ifdef _OPENMP
#include <omp.h>
#endif

__global__ void mmc_main_loop(const int nphoton, const int ophoton,
                              MCXParam *gcfg, float3 *node, int *elem,
                              float *weight, float *dref, int *type,
                              int *facenb, int *srcelem, float4 *normal,
                              medium *med, float4 *gdetpos, float *n_det,
                              uint *detectedphoton, uint *n_seed, int *progress,
                              float *energy, MCXReporter *reporter);

/************************************************************************** In
this unit, we first launch a master thread and initialize the necessary data
structures.This include the command line options(cfg), tetrahedral mesh(mesh)
and the ray tracer precomputed data (tracer).
******************************************************************************/
#define CUDA_ASSERT(a)      mcx_cu_assess((a),__FILE__,__LINE__) ///< macro to report CUDA error
int mcx_corecount(int v1, int v2) {
  int v = v1 * 10 + v2;
  if (v < 20)
    return 8;
  else if (v < 21)
    return 32;
  else if (v < 30)
    return 48;
  else if (v < 50)
    return 192;
  else
    return 128;
}

int mcx_smxblock(int v1, int v2) {
  int v = v1 * 10 + v2;
  if (v < 30)
    return 8;
  else if (v < 50)
    return 16;
  else
    return 32;
}

/**
  assert cuda memory allocation result
 */
void mcx_cu_assess(hipError_t cuerr, const char *file, const int linenum) {
  if (cuerr != hipSuccess) {
    mcx_error(-(int)cuerr, (char *)hipGetErrorString(cuerr), file, linenum);
  }
}

/*
   master driver code to run MC simulations
*/
int mcx_list_gpu(mcconfig *cfg, GPUInfo **info) {
#if __DEVICE_EMULATION__
  return 1;
#else
  int dev;
  int deviceCount, activedev = 0;

  CUDA_ASSERT(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
    MCX_FPRINTF(stderr,
                S_RED "ERROR: No CUDA-capable GPU device found\n" S_RESET);
    return 0;
  }
  *info = (GPUInfo *)calloc(deviceCount, sizeof(GPUInfo));
  if (cfg->gpuid && cfg->gpuid > deviceCount) {
    MCX_FPRINTF(stderr,
                S_RED "ERROR: Specified GPU ID is out of range\n" S_RESET);
    return 0;
  }

  // scan from the first device
  for (dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t dp;
    CUDA_ASSERT(hipGetDeviceProperties(&dp, dev));

    if (cfg->isgpuinfo == 3)
      activedev++;
    else if (cfg->deviceid[dev] == '1') {
      cfg->deviceid[dev] = '\0';
      cfg->deviceid[activedev] = dev + 1;
      activedev++;
    }

    strncpy((*info)[dev].name, dp.name, MAX_SESSION_LENGTH);
    (*info)[dev].id = dev + 1;
    (*info)[dev].devcount = deviceCount;
    (*info)[dev].major = dp.major;
    (*info)[dev].minor = dp.minor;
    (*info)[dev].globalmem = dp.totalGlobalMem;
    (*info)[dev].constmem = dp.totalConstMem;
    (*info)[dev].sharedmem = dp.sharedMemPerBlock;
    (*info)[dev].regcount = dp.regsPerBlock;
    (*info)[dev].clock = dp.clockRate;
    (*info)[dev].sm = dp.multiProcessorCount;
    (*info)[dev].core =
        dp.multiProcessorCount * mcx_corecount(dp.major, dp.minor);
    (*info)[dev].maxmpthread = dp.maxThreadsPerMultiProcessor;
    (*info)[dev].maxgate = cfg->maxgate;
    (*info)[dev].autoblock =
        (*info)[dev].maxmpthread / mcx_smxblock(dp.major, dp.minor);
    (*info)[dev].autothread = (*info)[dev].autoblock *
                              mcx_smxblock(dp.major, dp.minor) *
                              (*info)[dev].sm;

    if (strncmp(dp.name, "Device Emulation", 16)) {
      if (cfg->isgpuinfo) {
        MCX_FPRINTF(stdout, S_BLUE
                    "=============================   GPU Infomation  "
                    "================================\n" S_RESET);
        MCX_FPRINTF(stdout, "Device %d of %d:\t\t%s\n", (*info)[dev].id,
                    (*info)[dev].devcount, (*info)[dev].name);
        MCX_FPRINTF(stdout, "Compute Capability:\t%u.%u\n", (*info)[dev].major,
                    (*info)[dev].minor);
        MCX_FPRINTF(stdout,
                    "Global Memory:\t\t%u B\nConstant Memory:\t%u B\n"
                    "Shared Memory:\t\t%u B\nRegisters:\t\t%u\nClock "
                    "Speed:\t\t%.2f GHz\n",
                    (unsigned int)(*info)[dev].globalmem,
                    (unsigned int)(*info)[dev].constmem,
                    (unsigned int)(*info)[dev].sharedmem,
                    (unsigned int)(*info)[dev].regcount,
                    (*info)[dev].clock * 1e-6f);
#if CUDART_VERSION >= 2000
        MCX_FPRINTF(stdout, "Number of MPs:\t\t%u\nNumber of Cores:\t%u\n",
                    (*info)[dev].sm, (*info)[dev].core);
#endif
        MCX_FPRINTF(stdout, "SMX count:\t\t%u\n", (*info)[dev].sm);
      }
    }
  }

  if (cfg->isgpuinfo == 2 &&
      cfg->parentid == mpStandalone) {  // list GPU info only
    exit(0);
  }

  if (activedev < MAX_DEVICE) {
    cfg->deviceid[activedev] = '\0';
  }

  return activedev;
#endif
}

void mmc_run_cl(mcconfig *cfg, tetmesh *mesh, raytracer *tracer) {
  uint i, j, iter;
  float t, twindow0, twindow1;
  float fullload = 0.f;
  float *energy;

  uint detected = 0, workdev;
  int gpuid, threadid = 0;
  uint tic, tic0, tic1, toc = 0, fieldlen;
  int threadphoton, oddphotons;
  dim3 mcgrid, mcblock;
  int status = 0;

  uint totalcucore;

  float3 *gnode;
  int4 *gelem, *gfacenb;
  float4 *gnormal, *gdetpos;
  int *gtype, *gsrcelem;
  uint *gseed, *gdetected;
  volatile int *progress, *gprogress;
  float *gweight;
  float *gdref;
  float *gdetphoton;
  float *genergy;
  float *gsrcpattern;

  medium *gproperty;
  MCXParam *gparam;
  MCXReporter *greporter;
  uint meshlen = ((cfg->method == rtBLBadouelGrid) ? cfg->crop0.z : mesh->ne)
                 << cfg->nbuffer;  // use 4 copies to reduce racing

  float *field, *dref = NULL;

  uint *Pseed;
  float *Pdet;

  char opt[MAX_PATH_LENGTH] = {'\0'};
  uint detreclen = (2 + ((cfg->ismomentum) > 0)) * mesh->prop +
                   (cfg->issaveexit > 0) * 6 + 1;
  uint hostdetreclen = detreclen + 1;
  GPUInfo *gpu = NULL;
  float3 srcdir = make_float3(cfg->srcdir.x, cfg->srcdir.y, cfg->srcdir.z);
  MCXParam param = {cfg->srcpos,
                    srcdir,
                    cfg->tstart,
                    cfg->tend,
                    (uint)cfg->isreflect,
                    (uint)cfg->issavedet,
                    (uint)cfg->issaveexit,
                    (uint)cfg->ismomentum,
                    (uint)cfg->isatomic,
                    (uint)cfg->isspecular,
                    1.f / cfg->tstep,
                    cfg->minenergy,
                    cfg->maxdetphoton,
                    mesh->prop,
                    cfg->detnum,
                    (uint)cfg->voidtime,
                    (uint)cfg->srctype,
                    cfg->srcparam1,
                    cfg->srcparam2,
                    cfg->issaveref,
                    cfg->maxgate,
                    (uint)cfg->debuglevel,
                    detreclen,
                    cfg->outputtype,
                    mesh->elemlen,
                    cfg->mcmethod,
                    cfg->method,
                    1.f / cfg->unitinmm,
                    0.f,  // cfg->srcpos.w,
                    mesh->nn,
                    mesh->ne,
                    mesh->nf,
                    mesh->nmin,
                    cfg->nout,
                    cfg->roulettesize,
                    cfg->srcnum,
                    cfg->crop0,
                    mesh->srcelemlen,
                    cfg->bary0,
                    cfg->e0,
                    cfg->isextdet,
                    meshlen,
                    cfg->nbuffer,
                    ((1 << cfg->nbuffer) - 1)};

  MCXReporter reporter = {0.f};
  workdev = mcx_list_gpu(cfg, &gpu);

  if (workdev > MAX_DEVICE) workdev = MAX_DEVICE;
  if (workdev == 0)
    mcx_error(-99, (char *)("Unable to find devices!"), __FILE__, __LINE__);
#ifdef _OPENMP
  threadid = omp_get_thread_num();
#endif
  if (threadid < MAX_DEVICE && cfg->deviceid[threadid] == '\0') return;

  gpuid = cfg->deviceid[threadid] - 1;
  if (gpuid < 0) mcx_error(-1, "GPU ID must be non-zero", __FILE__, __LINE__);
  CUDA_ASSERT(hipSetDevice(gpuid));

  //#pragma omp master
  {
    if (cfg->exportfield == NULL) cfg->exportfield = mesh->weight;
    if (cfg->exportdetected == NULL)
      cfg->exportdetected =
          (float *)malloc(hostdetreclen * cfg->maxdetphoton * sizeof(float));

    cfg->energytot = 0.f;
    cfg->energyesc = 0.f;
    cfg->runtime = 0;
  }
  //#pragma omp barrier

  gpu[gpuid].autothread = cfg->nthread;
  gpu[gpuid].autoblock = cfg->nblocksize;
  gpu[gpuid].maxgate = cfg->maxgate;

  if (gpu[gpuid].autothread % gpu[gpuid].autoblock)
    gpu[gpuid].autothread =
        (gpu[gpuid].autothread / gpu[gpuid].autoblock) * gpu[gpuid].autoblock;
  if (gpu[gpuid].maxgate == 0 && meshlen > 0) {
    int needmem = meshlen + gpu[gpuid].autothread * sizeof(float4) * 4 +
                  sizeof(float) * cfg->maxdetphoton * hostdetreclen +
                  10 * 1024 * 1024; /*keep 10M for other things*/
    gpu[gpuid].maxgate = (gpu[gpuid].globalmem - needmem) / meshlen;
    gpu[gpuid].maxgate =
        MIN(((cfg->tend - cfg->tstart) / cfg->tstep + 0.5), gpu[gpuid].maxgate);
  }

  cfg->maxgate = (int)((cfg->tend - cfg->tstart) / cfg->tstep + 0.5);
  param.maxgate = cfg->maxgate;
  uint nflen = mesh->nf * cfg->maxgate;
  //#pragma omp master
  fullload = 0.f;
  for (i = 0; i < workdev; i++) fullload += cfg->workload[i];

  if (fullload < EPS) {
    for (i = 0; i < workdev; i++) cfg->workload[i] = gpu[i].core;
    fullload = totalcucore;
  }
  //#pragma omp barrier

  threadphoton = (int)(cfg->nphoton * cfg->workload[i] /
                       (fullload * gpu[gpuid].autothread * cfg->respin));
  oddphotons =
      (int)(cfg->nphoton * cfg->workload[i] / (fullload * cfg->respin) -
            threadphoton * gpu[gpuid].autothread);
  field = (float *)calloc(sizeof(float) * meshlen, cfg->maxgate);
  dref = (float *)calloc(sizeof(float) * mesh->nf, cfg->maxgate);
  Pdet = (float *)calloc(cfg->maxdetphoton * sizeof(float), hostdetreclen);

  mcgrid.x = gpu[gpuid].autothread / gpu[gpuid].autoblock;
  mcblock.x = gpu[gpuid].autoblock;
  fieldlen = meshlen * cfg->maxgate;

  if (cfg->seed > 0)
    srand(cfg->seed);
  else
    srand(time(0));

  // create gpu pointer
  // gnode,gelem,gtype,gfacenb,gsrcelem,gnormal,gdetpos,gproperty and copy the
  // data from cpu to gpu
  CUDA_ASSERT(hipMalloc((void **)&gnode, sizeof(float3) * (mesh->nn)));
  CUDA_ASSERT(hipMemcpy(gnode, mesh->node, sizeof(float3) * (mesh->nn),
                         hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gelem, sizeof(int4) * (mesh->ne)));
  CUDA_ASSERT(hipMemcpy(gelem, mesh->elem, sizeof(int4) * (mesh->ne),
                         hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gtype, sizeof(int) * (mesh->ne)));
  CUDA_ASSERT(hipMemcpy(gtype, mesh->type, sizeof(int) * (mesh->ne),
                         hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gfacenb, sizeof(int4) * (mesh->ne)));
  CUDA_ASSERT(hipMemcpy(gfacenb, mesh->facenb, sizeof(int4) * (mesh->ne),
                         hipMemcpyHostToDevice));

  if (mesh->srcelemlen > 0) {
    CUDA_ASSERT(
        hipMalloc((void **)&gsrcelem, sizeof(int) * (mesh->srcelemlen)));
    CUDA_ASSERT(hipMemcpy(gsrcelem, mesh->srcelem,
                           sizeof(int) * (mesh->srcelemlen),
                           hipMemcpyHostToDevice));
  } else
    gsrcelem = NULL;

  CUDA_ASSERT(hipMalloc((void **)&gnormal, sizeof(float4) * (mesh->ne) * 4));
  CUDA_ASSERT(hipMemcpy(gnormal, tracer->n, sizeof(float4) * (mesh->ne) * 4,
                         hipMemcpyHostToDevice));

  if (cfg->detpos && cfg->detnum) {
    CUDA_ASSERT(hipMalloc((void **)&gdetpos, sizeof(float4) * (cfg->detnum)));
    CUDA_ASSERT(hipMemcpy(gdetpos, cfg->detpos, sizeof(float4) * (cfg->detnum),
                           hipMemcpyHostToDevice));
  } else {
    gdetpos = NULL;
  }

  CUDA_ASSERT(hipMalloc((void **)&gproperty,
                         (mesh->prop + 1 + cfg->isextdet) * sizeof(medium)));
  CUDA_ASSERT(hipMemcpy(gproperty, mesh->med,
                         (mesh->prop + 1 + cfg->isextdet) * sizeof(medium),
                         hipMemcpyHostToDevice));
  // gparam
  CUDA_ASSERT(hipMalloc((void **)&gparam, sizeof(MCXParam)));
  CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gparam), &param, sizeof(MCXParam), 0,
                                 hipMemcpyHostToDevice));
  // gprogress
  CUDA_ASSERT(
      hipHostAlloc((void **)&progress, sizeof(int), hipHostMallocMapped));
  CUDA_ASSERT(hipHostGetDevicePointer((int **)&gprogress, (int *)progress, 0));
  *progress = 0;

  Pseed =
      (uint *)malloc(sizeof(int) * gpu[gpuid].autothread * RAND_SEED_WORD_LEN);
  energy = (float *)calloc(sizeof(float), gpu[gpuid].autothread << 1);
  for (j = 0; j < gpu[i].autothread * RAND_SEED_WORD_LEN; j++)
    Pseed[j] = rand();

  CUDA_ASSERT(hipMalloc((void **)&gseed, sizeof(uint) * gpu[gpuid].autothread *
                                              RAND_SEED_WORD_LEN));
  CUDA_ASSERT(hipMemcpy(
      gseed, Pseed, sizeof(uint) * gpu[gpuid].autothread * RAND_SEED_WORD_LEN,
      hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gweight, sizeof(float) * fieldlen));
  CUDA_ASSERT(hipMemcpy(gweight, field, sizeof(float) * fieldlen,
                         hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gdref, sizeof(float) * nflen));
  CUDA_ASSERT(
      hipMemcpy(gdref, dref, sizeof(float) * nflen, hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gdetphoton,
                         sizeof(float) * cfg->maxdetphoton * hostdetreclen));
  CUDA_ASSERT(hipMemcpy(gdetphoton, Pdet,
                         sizeof(float) * cfg->maxdetphoton * hostdetreclen,
                         hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&genergy,
                         sizeof(float) * (gpu[gpuid].autothread << 1)));
  CUDA_ASSERT(hipMemcpy(genergy, energy,
                         sizeof(float) * (gpu[gpuid].autothread << 1),
                         hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&gdetected, sizeof(uint)));
  CUDA_ASSERT(
      hipMemcpy(gdetected, &detected, sizeof(uint), hipMemcpyHostToDevice));

  CUDA_ASSERT(hipMalloc((void **)&greporter, sizeof(MCXReporter)));
  CUDA_ASSERT(hipMemcpy(greporter, &reporter, sizeof(MCXReporter),
                         hipMemcpyHostToDevice));

  if (cfg->srctype == MCX_SRC_PATTERN) {
    CUDA_ASSERT(
        hipMalloc((void **)&gsrcpattern,
                   sizeof(float) * (int)(cfg->srcparam1.w * cfg->srcparam2.w)));
    CUDA_ASSERT(
        hipMemcpy(gsrcpattern, cfg->srcpattern,
                   sizeof(float) * (int)(cfg->srcparam1.w * cfg->srcparam2.w),
                   hipMemcpyHostToDevice));
  } else if (cfg->srctype == MCX_SRC_PATTERN3D) {
    CUDA_ASSERT(hipMalloc(
        (void **)&gsrcpattern,
        sizeof(float) *
            (int)(cfg->srcparam1.x * cfg->srcparam1.y * cfg->srcparam1.z)));
    CUDA_ASSERT(hipMemcpy(
        gsrcpattern, cfg->srcpattern,
        sizeof(float) *
            (int)(cfg->srcparam1.x * cfg->srcparam1.y * cfg->srcparam1.z),
        hipMemcpyHostToDevice));
  } else {
    gsrcpattern = NULL;
  }

  free(Pseed);
  free(energy);
  tic = StartTimer();

  //#pragma omp master
  {
    mcx_printheader(cfg);

#ifdef MCX_TARGET_NAME
    MCX_FPRINTF(
        cfg->flog,
        "- variant name: [%s] compiled by nvcc [%d.%d] with CUDA [%d]\n",
        "Fermi", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, CUDART_VERSION);
#else
    MCX_FPRINTF(
        cfg->flog,
        "- code name: [Vanilla MCX] compiled by nvcc [%d.%d] with CUDA [%d]\n",
        __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, CUDART_VERSION);
#endif
    MCX_FPRINTF(cfg->flog, "- compiled with: [RNG] %s [Seed Length] %d\n",
                MCX_RNG_NAME, RAND_SEED_WORD_LEN);
    fflush(cfg->flog);
  }
  //#pragma omp barrier

  MCX_FPRINTF(cfg->flog,
              "- [device %d(%d): %s] threadph=%d oddphotons=%d np=%.1f "
              "nthread=%d nblock=%d repetition=%d\n",
              gpuid + 1, gpu[gpuid].id, gpu[gpuid].name, threadphoton,
              oddphotons, cfg->nphoton * cfg->workload[gpuid] / fullload,
              (int)gpu[gpuid].autothread, (int)gpu[gpuid].autoblock,
              cfg->respin);

  // simulate for all time-gates in maxgate groups per run

  tic0 = GetTimeMillis();

  for (t = cfg->tstart; t < cfg->tend; t += cfg->tstep * cfg->maxgate) {
    twindow0 = t;
    twindow1 = t + cfg->tstep * cfg->maxgate;

    MCX_FPRINTF(cfg->flog,
                "lauching mcx_main_loop for time window [%.1fns %.1fns] ...\n",
                twindow0 * 1e9, twindow1 * 1e9);
    fflush(cfg->flog);

    // total number of repetition for the simulations, results will be
    // accumulated to field
    for (iter = 0; iter < cfg->respin; iter++) {
      MCX_FPRINTF(cfg->flog, "simulation run#%2d ... \n", iter + 1);
      fflush(cfg->flog);
      fflush(cfg->flog);
      param.tstart = twindow0;
      param.tend = twindow1;

      // launch mcxkernel
      size_t sharedMemSize = sizeof(int);
      if (cfg->issavedet) {
        sharedMemSize = sizeof(float) * ((int)gpu[i].autoblock) * detreclen;
      }
      mmc_main_loop<<<mcgrid, mcblock, sharedMemSize>>>(
          threadphoton, 
          oddphotons, 
          gparam, 
          gnode, 
          (int*)gelem, 
          gweight, 
          gdref, 
          gtype,
          (int *)gfacenb, 
          gsrcelem, 
          gnormal, 
          gproperty, 
          gdetpos, 
          gdetphoton,
          gdetected, 
          gseed,
          (int *)gprogress, 
          genergy, 
          greporter);

      //#pragma omp master
      {
        if ((cfg->debuglevel & MCX_DEBUG_PROGRESS)) {
          int p0 = 0, ndone = -1;

          mcx_progressbar(-0.f, cfg);

          do {
            ndone = *progress;

            if (ndone > p0) {
              mcx_progressbar((float)ndone / gpu[0].autothread * cfg->nphoton,
                              cfg);
              p0 = ndone;
            }
            sleep_ms(100);
          } while (p0 < gpu[0].autothread);
          mcx_progressbar(cfg->nphoton, cfg);
          MCX_FPRINTF(cfg->flog, "\n");
        }
      }
      CUDA_ASSERT(hipDeviceSynchronize());
      tic1 = GetTimeMillis();
      toc += tic1 - tic0;
      MCX_FPRINTF(cfg->flog,
                  "kernel complete:  \t%d ms\nretrieving flux ... \t",
                  tic1 - tic);
      fflush(cfg->flog);
      //#pragma omp critical
      if (cfg->runtime < tic1 - tic) cfg->runtime = tic1 - tic;

      MCXReporter rep;
      CUDA_ASSERT(hipMemcpy(&rep, greporter, sizeof(MCXReporter),
                             hipMemcpyDeviceToHost));
      reporter.raytet += rep.raytet;
      if (cfg->issavedet) {
        CUDA_ASSERT(hipMemcpy(&detected, gdetected, sizeof(uint),
                               hipMemcpyDeviceToHost));

        CUDA_ASSERT(
            hipMemcpy(Pdet, gdetphoton,
                       sizeof(float) * cfg->maxdetphoton * hostdetreclen,
                       hipMemcpyDeviceToHost));
        if (detected > cfg->maxdetphoton) {
          MCX_FPRINTF(cfg->flog,
                      "WARNING: the detected photon (%d) \
              is more than what your have specified (%d), please use the -H option to specify a greater number\t",
                      detected, cfg->maxdetphoton);
        } else {
          MCX_FPRINTF(cfg->flog, "detected %d photons, total: %d\t", detected,
                      cfg->detectedcount + detected);
        }
        //#pragma omp atomic
        cfg->his.detected += detected;
        detected = MIN(detected, cfg->maxdetphoton);
        if (cfg->exportdetected) {
          //#pragma omp critical
          {
            cfg->exportdetected = (float *)realloc(
                cfg->exportdetected, (cfg->detectedcount + detected) *
                                         hostdetreclen * sizeof(float));
            memcpy(cfg->exportdetected + cfg->detectedcount * (hostdetreclen),
                   Pdet, detected * (hostdetreclen) * sizeof(float));
            cfg->detectedcount += detected;
          }
        }
      }
      if (cfg->issaveref) {
        float *rawdref = (float *)calloc(sizeof(float), nflen);

        CUDA_ASSERT(hipMemcpy(rawdref, gdref, sizeof(float) * nflen,
                               hipMemcpyDeviceToHost));
        for (i = 0; i < nflen; i++)  // accumulate field, can be done in the GPU
          dref[i] += rawdref[i];     //+rawfield[i+fieldlen];
        free(rawdref);
      }
      // handling the 2pt distributions
      if (cfg->issave2pt) {
        float *rawfield = (float *)malloc(sizeof(float) * fieldlen);

        CUDA_ASSERT(hipMemcpy(rawfield, gweight,
                               sizeof(float) * fieldlen,
                               hipMemcpyDeviceToHost));
        MCX_FPRINTF(cfg->flog, "transfer complete:        %d ms\n",
                    GetTimeMillis() - tic);
        fflush(cfg->flog);

        for (i = 0; i < fieldlen;
             i++)  // accumulate field, can be done in the GPU
          field[(i >> cfg->nbuffer)] += rawfield[i];  //+rawfield[i+fieldlen];

        free(rawfield);

        /*        	if(cfg->respin>1){
                    for(i=0;i<fieldlen;i++)  //accumulate field, can be done in
           the GPU field[fieldlen+i]+=field[i];
                    }
                    if(iter+1==cfg->respin){
                    if(cfg->respin>1)  //copy the accumulated fields back
                    memcpy(field,field+fieldlen,sizeof(cl_float)*fieldlen);
                    }
         */
        if (cfg->isnormalized) {
          energy = (float*)calloc(sizeof(float), gpu[gpuid].autothread << 1);

          CUDA_ASSERT(hipMemcpy(energy, genergy,
                                 sizeof(float) * (gpu[gpuid].autothread << 1),
                                 hipMemcpyDeviceToHost));
          //#pragma omp critical
          {
            for (i = 0; i < gpu[gpuid].autothread; i++) {
              cfg->energyesc += energy[(i << 1)];
              cfg->energytot += energy[(i << 1) + 1];
              // eabsorp+=Plen[i].z;  // the accumulative absorpted energy near
              // the source
            }
          }
          free(energy);
        }
      }
      if (cfg->respin > 1 && RAND_SEED_WORD_LEN > 1) {
        Pseed =
            (uint*)malloc(sizeof(uint) * gpu[gpuid].autothread * RAND_SEED_WORD_LEN);
        for (i = 0; i < gpu[gpuid].autothread * RAND_SEED_WORD_LEN; i++)
          Pseed[i] = rand();
        CUDA_ASSERT(hipMemcpy(
            gseed, Pseed,
            sizeof(uint) * gpu[gpuid].autothread * RAND_SEED_WORD_LEN,
            hipMemcpyHostToDevice));
        free(Pseed);
      }

      // loop over work devices
    }  // iteration
  }    // time gates
  //#pragma omp master
  {
    fieldlen = (fieldlen >> cfg->nbuffer);
    field = (float*)realloc(field, sizeof(field[0]) * fieldlen);
    if (cfg->exportfield) {
      if (cfg->basisorder == 0 || cfg->method == rtBLBadouelGrid) {
        for (i = 0; i < fieldlen; i++)
          //#pragma omp atomic
          cfg->exportfield[i] += field[i];
      } else {
        for (i = 0; i < cfg->maxgate; i++)
          for (j = 0; j < mesh->ne; j++) {
            float ww = field[i * mesh->ne + j] * 0.25f;
            int k;
            for (k = 0; k < mesh->elemlen; k++)
              cfg->exportfield[i * mesh->nn +
                               mesh->elem[j * mesh->elemlen + k] - 1] += ww;
          }
      }
    }

    if (cfg->issaveref && mesh->dref) {
      for (i = 0; i < nflen; i++) mesh->dref[i] += dref[i];
    }

    if (cfg->isnormalized) {
      MCX_FPRINTF(cfg->flog, "normalizing raw data ...\t");
      fflush(cfg->flog);

      cfg->energyabs = cfg->energytot - cfg->energyesc;
      mesh_normalize(mesh, cfg, cfg->energyabs, cfg->energytot, 0);
    }
    if (cfg->issave2pt && cfg->parentid == mpStandalone) {
      MCX_FPRINTF(cfg->flog, "saving data to file ...\t");
      mesh_saveweight(mesh, cfg, 0);
      MCX_FPRINTF(cfg->flog, "saving data complete : %d ms\n\n",
                  GetTimeMillis() - tic);
      fflush(cfg->flog);
    }
    if (cfg->issavedet && cfg->parentid == mpStandalone &&
        cfg->exportdetected) {
      cfg->his.unitinmm = cfg->unitinmm;
      cfg->his.savedphoton = cfg->detectedcount;
      cfg->his.detected = cfg->detectedcount;
      mesh_savedetphoton(cfg->exportdetected, NULL, cfg->detectedcount,
                         (sizeof(uint64_t) * RAND_BUF_LEN), cfg);
    }
    if (cfg->issaveref) {
      MCX_FPRINTF(cfg->flog, "saving surface diffuse reflectance ...");
      mesh_saveweight(mesh, cfg, 1);
    }
    // total energy here equals total simulated photons+unfinished photons for
    // all threads
    MCX_FPRINTF(cfg->flog,
                "simulated %ld photons (%ld) with %d devices (ray-tet "
                "%.0f)\nMCX simulation speed: %.2f photon/ms\n",
                cfg->nphoton, cfg->nphoton, workdev, reporter.raytet,
                (double)cfg->nphoton / toc);
    MCX_FPRINTF(cfg->flog,
                "total simulated energy: %.2f\tabsorbed: %5.5f%%\n(loss due to "
                "initial specular reflection is excluded in the total)\n",
                cfg->energytot,
                (cfg->energytot - cfg->energyesc) / cfg->energytot * 100.f);
    fflush(cfg->flog);
  }
  //#pragma omp barrier
  hipFree(gnode);
  hipFree(gelem);
  hipFree(gtype);
  hipFree(gfacenb);
  hipFree(gsrcelem);
  hipFree(gnormal);
  hipFree(gproperty);
  hipFree(gparam);
  if (cfg->detpos) hipFree(gdetpos);

  hipFree(gseed);
  hipFree(gdetphoton);
  hipFree(gweight);
  hipFree(gdref);
  hipFree(genergy);
  hipFree((int*)gprogress);
  hipFree(gdetected);
  if (gsrcpattern) hipFree(gsrcpattern);
  hipFree(greporter);

  if (gpu) free(gpu);

  free(field);
  if (Pdet) free(Pdet);
  free(dref);
}
